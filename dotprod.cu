
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


// P = max power of 2 to test up to
// i.e., test for N = 2^0, 2^1, 2^2... 2^P
#define P 8
#define ThreadsPerBlock (1<<10)
#define MAX_TILE_WIDTH 16
#define VERBOSE 0

__global__ void dot(float* a, float* b, float* c, unsigned int width) {
    __shared__ float temp[ThreadsPerBlock];
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int index = ThreadsPerBlock*bx + tx;
    int sumrange = width < ThreadsPerBlock ? width : ThreadsPerBlock;

    if(index < width) {
        temp[tx] = a[index]*b[index];
    }

    __syncthreads();
    // Iterative halving sum
    for(int offset = sumrange >> 1; offset > 0; offset >>= 1) {
        if(tx < offset) {
            temp[tx] += temp[tx+offset];
        }
        __syncthreads();
    }

    if(tx == 0) {
        c[bx] = temp[0];
    }

}

__global__ void matrixMultKernel(float* Md, float* Nd, float* Pd, int Width, int tile_width) {
    // Notice that we are allocating MORE shared memory than we
    // will actually use.  MAX_TILE_WIDTH^2 (each) is allocated
    // but only tile_width^2 (each) is used for computation.
    __shared__ float Mds[MAX_TILE_WIDTH][MAX_TILE_WIDTH];
    __shared__ float Nds[MAX_TILE_WIDTH][MAX_TILE_WIDTH];
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    // Identify the row and column of the Pd element to work on
    int Row = by * tile_width + ty;
    int Col = bx * tile_width + tx;

    float Pvalue = 0;
    // Loop over the Md and Nd tiles required to compute the Pd element

    for (int m = 0; m < Width/tile_width; ++m) {
        // Collaborative loading of Md and Nd tiles into shared memory
        Mds[ty][tx] = Md[Row*Width + (m*tile_width + tx)];
        Nds[ty][tx] = Nd[Col + (m*tile_width + ty)*Width];
        __syncthreads();

        for (int k = 0; k < tile_width; ++k)
            Pvalue += Mds[ty][k] * Nds[k][tx];
        __syncthreads();
    }
    Pd[Row*Width+Col] = Pvalue;
}

// Num subresults is the number of sub- dot products computed in the
// GPU.  The host will add them all up.
float dotprod(float* a, float* b, unsigned int width) {
    unsigned int size_C; // Number of elements in result vector
    unsigned int mem_size_C;
    float ret;
    float* h_C;
    float* d_A;
    float* d_B;
    float* d_C;

    // Allocate device memory for vectors A and B
    unsigned int mem_size_Vect = sizeof(float) * width;
    hipMalloc((void**) &d_A, mem_size_Vect);
    hipMalloc((void**) &d_B, mem_size_Vect);

    // Copy host memory to device
    hipMemcpy(d_A, a, mem_size_Vect, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, mem_size_Vect, hipMemcpyHostToDevice);

    // Allocate host memory for the result C = A dot B
    size_C = 1 + ((width - 1) / ThreadsPerBlock);
    mem_size_C = sizeof(float) * size_C;
    h_C = (float*) malloc(mem_size_C);
    *h_C = 0;

    // Allocate device memory for the result
    hipMalloc((void**) &d_C, mem_size_C);

    // Set up the calculation
    dim3 blocks_Vect(ThreadsPerBlock);
    dim3 grid_Vect(size_C);

    dot<<< grid_Vect, blocks_Vect >>>(d_A, d_B, d_C, width);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    // Finish adding together the partial sums on the host (linearly).
    // See the kernel dot product function to see the iterative halving
    // (i.e., O(log n)) sum.

    for(int i = 1; i < size_C; i++) {
        h_C[0] += h_C[i];
    }

    ret = h_C[0];

    // Clean up memory
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return ret;
}

// Multiplies A with B and puts the result in C
void matrixMult(float* A, float* B, float* C, int width, int tile_width) {
    // Memory allocation grunt work
    unsigned int mem_size_Matrix = sizeof(float) * width * width;
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**) &d_A, mem_size_Matrix);
    hipMalloc((void**) &d_B, mem_size_Matrix);
    hipMalloc((void**) &d_C, mem_size_Matrix);
    hipMemcpy(d_A, A, mem_size_Matrix, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, mem_size_Matrix, hipMemcpyHostToDevice);

    // Set up and perform the actual computation
    dim3 blocks(tile_width, tile_width);
    dim3 grid(width / tile_width, width / tile_width);

    matrixMultKernel<<< grid, blocks, 2*tile_width*tile_width >>> (
            d_A, d_B, d_C, width, tile_width
    );

    // Copy result from device to host
    hipMemcpy(C, d_C, mem_size_Matrix, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)(RAND_MAX+1.0);
}

// Copies a random row from M to V
void extractRow(float* V, float* M, int rowlen, int row) {
    for(int i = 0; i < rowlen; i++) {
        V[i] = M[row*rowlen + i];
    }
}

// Copies a random column from M to V
void extractCol(float* V, float* M, int collen, int col) {
    for(int i = 0; i < collen; i++) {
        V[i] = M[i*collen + col];
    }
}

int main(int argc, char** argv) {

    unsigned int size_Vect; // Number of elements in vectors
    unsigned int mem_size_Vect;
    unsigned int size_Matrix; // Number of elements in matricies
    unsigned int mem_size_Matrix;

    float dotprod_expected; // Computed by dotprod -- not by Matrix Mult
    float dotprod_ABij; // Value of AB at row i column j (random sample)
    int random_i; // Random i to choose a row
    int random_j; // Random j to choose a column

    float* h_Row; // Vectors for a Row and Column
    float* h_Col;
    float* h_A; // Matricies A and B
    float* h_B;
    float* h_C; // Matrix multiplication AB result

    // Seed the random number generator
    // Let's use this year for fun
    srand(2015);

    // Test for different powers
    for(int p = 1; p <= P; p++) {
        printf("p=%d (N=2^%d)\n", p, p);

        // Allocate host memory fors vector Row and Col
        size_Vect = 1<<p;
        mem_size_Vect = sizeof(float) * size_Vect;
        h_Row = (float*) malloc(mem_size_Vect);
        h_Col = (float*) malloc(mem_size_Vect);

        // Allocate host memory for matricies A and B
        size_Matrix = size_Vect * size_Vect;
        mem_size_Matrix = sizeof(float) * size_Matrix;
        h_A = (float*) malloc(mem_size_Matrix);
        h_B = (float*) malloc(mem_size_Matrix);
        h_C = (float*) malloc(mem_size_Matrix);
        memset(h_C, 0, mem_size_Matrix);
        
        // Initialize host memory for matricies Row and Col
        randomInit(h_A, size_Matrix);
        randomInit(h_B, size_Matrix);

        // Initialize host memory for vectors Row and Col
        // These are random samples
        random_i = (int)(rand() % size_Vect);
        random_j = (int)(rand() % size_Vect);
        extractRow(h_Row, h_A, size_Vect, random_i);
        extractCol(h_Col, h_B, size_Vect, random_j);

        // Perform the dot product
        dotprod_expected = dotprod(h_Row, h_Col, size_Vect);

        printf("    (row i, col j) = (%d, %d)\n", random_i, random_j);
        printf("    Expected dot product   = %0.5f...\n",
                dotprod_expected);

        #if VERBOSE
            printf("    Row i = <  ");
            for (int i=0; i < size_Vect; i++) {
                        printf("%0.5f  ", h_Row[i]);
            }
            printf(">\n");
            printf("    Col j = <  ");
            for (int i=0; i < size_Vect; i++) {
                printf("%0.5f  ", h_Col[i]);
            }
            printf(">\n");
        #endif
        for(int tile_width = 1; tile_width <= MAX_TILE_WIDTH; tile_width <<= 1) {
            // Don't test tiles that are larger than the respective matricies
            if(size_Vect < tile_width) { break; }

            // Perform the matrix multiplication
            memset(h_C, 0, mem_size_Matrix);
            matrixMult(h_A, h_B, h_C, size_Vect, tile_width);

            // Extract the desired dot product
            dotprod_ABij = h_C[size_Vect*random_i + random_j];

            // Print results
            printf("    tile_width = %d\n", tile_width);
            printf("        MM dot product = %0.5f...\n",
                    dotprod_ABij);
            #if VERBOSE
                printf("\n");
                for (int i=0; i < size_Matrix; i++) {
                    if(i % size_Vect > 0) { printf(" "); }
                    printf("        %0.5f : %0.5f  :  %0.5f %s \n", h_A[i], h_B[i], h_C[i],
                            i == size_Vect*random_i + random_j ? "*" : "");
                }
            #endif
        }

        // Clean up memory
        free(h_Row);
        free(h_Col);
        free(h_A);
        free(h_B);
        free(h_C);
    }
}