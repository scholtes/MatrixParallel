
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


// P = max power of 2 to test up to
// i.e., test for N = 2^0, 2^1, 2^2... 2^P
#define P 6
#define TILE_WIDTH 1
#define ThreadsPerBlock (1<<3)
#define BlocksPerGrid ((1<<16)-1)
#define RANDRANGE  5
#define VERBOSE 0

__global__ void dot(float* a, float* b, float* c, unsigned int width) {
    __shared__ float temp[ThreadsPerBlock];
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int index = ThreadsPerBlock*bx + tx;

    if(index < width) {
        temp[tx] = a[index]*b[index];
    }
    //__syncthreads();
    if(tx == 0) {
        float sum = 0;
        for(int i = 0; i < width; i++) {
            sum += temp[i];
        }
        c[bx] = sum;
    }
}

// Num subresults is the number of sub- dot products computed in the
// GPU.  The host will add them all up.
void dotprod(float* a, float* b, float* c, unsigned int width,
        unsigned int num_subresults) {
    // Set up the calculation
    dim3 blocks_Vect(ThreadsPerBlock);
    dim3 grid_Vect(num_subresults);

    dot<<< grid_Vect, blocks_Vect >>>(a, b, c, width);

    // Perform additional sums on the host
    //for(int i = 1; i < num_subresults; i++) {
    //    c[0] += c[i];
    //}
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i)
        data[i] = (float)(rand() % RANDRANGE +1);
}

int main(int argc, char** argv) {

    unsigned int size_Vect; // Number of elements in vectors
    unsigned int size_C; // Number of elements in result vector
    unsigned int mem_size_Vect;
    unsigned int mem_size_C;
    float* h_A;
    float* h_B;
    float* h_C;
    float* d_A;
    float* d_B;
    float* d_C;

    // Test for different powers
    for(int p = 1; p <= P; p++) {

        // Allocate host memory fors vector A and B
        size_Vect = 1<<p;
        mem_size_Vect = sizeof(float) * size_Vect;
        h_A = (float*) malloc(mem_size_Vect);
        h_B = (float*) malloc(mem_size_Vect);

        // Initialize host memory for vectors A and B
        // We seed twice so that the beginning sequences in the
        // loop are the same
        srand(0);
        randomInit(h_A, size_Vect);
        srand(1);
        randomInit(h_B, size_Vect);

        // Allocate device memory for vectors A and B
        hipMalloc((void**) &d_A, mem_size_Vect);
        hipMalloc((void**) &d_B, mem_size_Vect);

        // Copy host memory to device
        hipMemcpy(d_A, h_A, mem_size_Vect, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, mem_size_Vect, hipMemcpyHostToDevice);

        // Allocate host memory for the result C = A dot B
        size_C = 1 + size_Vect/ThreadsPerBlock;
        mem_size_C = sizeof(float) * size_C;
        h_C = (float*) malloc(mem_size_C);
        *h_C = 0;

        // Allocate device memory for the result
        hipMalloc((void**) &d_C, mem_size_C);

        // Perform the calculation
        dotprod(d_A, d_B, d_C, size_Vect, size_C);
        
        
        // Copy result from device to host
        // We only care about the first element, so we only copy that
        hipMemcpy(h_C, d_C, sizeof(float), hipMemcpyDeviceToHost);

        // Basic test
        #if VERBOSE
            printf("A = [ ");
            for(int i=0; i < size_Vect; i++) {
                printf("%0.1f ", h_A[i]);
            }
            printf("]\nB = [ ");
            for(int i=0; i < size_Vect; i++) {
                printf("%0.1f ", h_B[i]);
            }
            printf("]\n");
        #endif
        printf("C = %0.2f\n", *h_C);

        // Clean up memory
        free(h_A);
        free(h_B);
        free(h_C);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
}